
#include <hip/hip_runtime.h>
#define SWAP(a, b) do { float __tmp = a; a = b; b = __tmp; } while(0)
#define SWAP_ROW(a, b, cnt)  do { for (int __i = 0; __i < cnt; __i++)  { SWAP(a[__i], b[__i]); } } while(0)
#define COPY_VEC(a, b, cnt)  do { for (int __iii = 0; __iii < cnt; __iii++) { b[__iii] = a[__iii]; } } while(0)
#define ELEM_WISE_VEC_VEC(a, b, cnt) do { for (int __ii = 0; __ii < cnt; __ii++) { a[__ii] -= b[__ii]; } } while(0)
#define ELEM_WISE_VEC_SCALAR(a, b, cnt) do { for (int __j = 0; __j < cnt; __j++ ) { a[__j] *= b; } } while(0)

// P = L = I
// U = A; main matrix
// A = [count][count] square matrix
extern "C" __global__ void lu_decompose_pivot(float** L, float** U, float** P, int count) {
    for (int t = blockIdx.x * blockDim.x + threadIdx.x; t < count; t += blockDim.x * gridDim.x) {
        for (int i = 0; i < t; i++) {
            int k = i;

            while (U[i][i] == 0) {
                SWAP_ROW(U[i], U[k + 1], count);
                SWAP_ROW(P[i], P[k + 1], count);

                ++k;
            }

            for (int j = i + 1; j < t; j++) {
                L[j][i] = U[j][i] / U[i][i];

                float cpy[count];
                COPY_VEC(U[i], cpy, count);

                ELEM_WISE_VEC_SCALAR(cpy, L[j][i], count);
                ELEM_WISE_VEC_VEC(U[j], cpy, count);
            }


        }
    }
}